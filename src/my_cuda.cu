#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hipblas.h"
extern "C" {
#include "my_cuda.h"
#include <cstdio>
#include <omp.h>
}

// __global__
// void saxpy(int n, float a, float * restrict x, float * restrict y)
// {
//   int i = blockIdx.x*blockDim.x + threadIdx.x;
//   if (i < n) y[i] = a*x[i] + y[i];
// }



extern "C"
{
void calculate_manhattan_cuda(float value1, int num_values2, int idx_value2, float *value2, 
int num_attr,float min, float max, float *dissim)
{

//   int dev = findCudaDevice(argc, (const char **)argv);

//   if (dev == -1) {
//     printf("No CUDA device found");
//   }
hipError_t status;
status = hipSetDevice(0);
if (status != hipSuccess)
{
   fprintf(stderr, "!!!! CUDA initialization error\n"); 
}

// int N = num_values2*num_attr;
// #pragma omp single
// #pragma omp criticall
{
    // printf("kernel\n");
hipblasHandle_t handle;
hipblasStatus_t cbstatus = hipblasCreate(&handle);

  if (cbstatus != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr, "!!!! CUBLAS initialization error\n");
  }

    float temp[num_values2];
    for (int i = 0; i < num_values2; i++) 
    {
        temp[i] = value1;
    }
float *x, *y, alpha=-1.0;
hipMalloc(&x, num_values2 * sizeof(float));
hipMalloc(&y, num_values2 * sizeof(float));
hipblasSaxpy(handle, num_values2, &alpha, x, 1, y, 1);
// hipblasDestroy(handle);
hipFree(x);
hipFree(y);
}
//     cblas_scopy(num_values2, &value1, 0, temp, 1);
//     cblas_saxpy(num_values2, -1, &value2[idx_value2], num_attr, temp, 1);

// // Perform SAXPY on 1M elements
// saxpy<<<4096,256>>>(N, 2.0, d_x, d_y);

// hipMemcpy(y, d_y, N, hipMemcpyDeviceToHost);





    //     float temp ;
    // for (int i = 0; i < num_values2; i++) 
    // {
        
    //     temp = value2[idx_value2*num_values2 + i] - value1;
    //     temp = temp > 0 ? temp : -temp;
    //     dissim[i] += temp / (max - min);
    // }



    // printf("kernel\n");
    // float temp[num_values2];
    // float *temp = malloc(sizeof(*temp) * num_values2);
    // float temp ;
    // for (int i = 0; i < num_values2; i++) 
    // {
    //     // printf("%d\n", i);
    //     temp = value1 - value2[idx_value2 + num_attr*i];
    //     temp = temp > 0 ? temp : -temp;
    //     temp = 1 / (max - min);
    //     dissim[i] += temp;
    // }

    // free(temp);
}


}

// void calculate_manhattan_vect(float value1, int num_values2, int idx_value2, float *value2, 
// int num_attr,float min, 
// float max, float *dissim)
// {

    // float temp[num_values2];
    // for (int i = 0; i < num_values2; i++) 
    // {
    //     temp[i] = value1 - value2[idx_value2 + num_attr*i];
    //     temp[i] = temp[i] > 0 ? temp[i] : -temp[i];
    //     temp[i] = 1 / (max - min);
    //     dissim[i] += temp[i];
    // }
// ///////////////////////////////

//     float temp[num_values2];
//     // for (int i = 0; i < num_values2; i++) 
//     // {
//     //     temp[i] = value1;
//     // }
//     cblas_scopy(num_values2, &value1, 0, temp, 1);
//     cblas_saxpy(num_values2, -1, &value2[idx_value2], num_attr, temp, 1);

//     for (int i = 0; i < num_values2; i++) 
//     {
//         temp[i] = temp[i] > 0 ? temp[i] : -temp[i];
//     }
//     // cblas_scabs1(num_values2, 1, temp, 1, dissim, 1);
//     cblas_sscal(num_values2, 1 / (max - min), temp, 1);
//     // cblas_saxpy(num_values2, 1, temp, 1, dissim, 1);
//     for (int i = 0; i < num_values2; i++) 
//     {
//         dissim[i] += temp[i];
//     }
// }