#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hipblas.h"

extern "C" {
#include "my_cuda.h"
#include <cstdio>
#include <omp.h>
}

#define NUM_THREADS 6

hipblasHandle_t handles[NUM_THREADS];

__global__ void saxpy_kernel(int n, float a, float * x, float * y){
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

__global__ void manhattan_kernel(int num_values2, float * values2, 
  float value1, float min, float max, float *dissim){
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  float temp;
  if (i < num_values2) 
  {
        temp = values2[i] - value1;
        temp = temp > 0 ? temp : -temp;
        dissim[i] += temp / (max - min);
  }
}

void my_cuda_init(int id)
{
  hipblasStatus_t cbstatus = hipblasCreate(&handles[id]);

  if (cbstatus != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr, "!!!! CUBLAS initialization error\n");
  }
}

void my_cuda_destroy(int id)
{
  hipblasStatus_t cbstatus = hipblasDestroy(handles[id]);
  if (cbstatus != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr, "!!!! CUBLAS destroy error\n");
  }
}



float *d_train, *d_test;
float *d_temp[NUM_THREADS];
int num_train, num_test, num_attr;

void copy_train_test_cuda(int num_train, int num_test, int num_attr, float *train,
    float *test)
{
  int n_bytes = num_train*num_attr*sizeof(train[0]);
  hipMalloc(&d_train, n_bytes);
  hipMemcpy(d_train, train, n_bytes, hipMemcpyHostToDevice);

  n_bytes = num_train * sizeof(float);
    for (int i = 0; i < NUM_THREADS; i++) 
    {
      hipMalloc(&d_temp[i], n_bytes);
    }

}

void free_test_train_cuda()
{
  hipFree(d_train);

  for (int i = 0; i < NUM_THREADS; i++) 
  {
    hipFree(&d_temp[i]);
  }
}

extern "C"
{


void calculate_manhattan_cuda(int id, float value1, int num_values2, int idx_value2, float *values2, 
int num_attr,float min, float max, float *dissim)
{
     size_t n_bytes = num_values2 * sizeof(float);

    hipMemcpy(d_temp[id], dissim, n_bytes, hipMemcpyHostToDevice);

    manhattan_kernel<<<4096,32>>>(num_values2, &d_train[idx_value2*num_values2], 
      value1, min, max, d_temp[id]);


      hipMemcpy(dissim, d_temp[id], n_bytes, hipMemcpyDeviceToHost);
}
}